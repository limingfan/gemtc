
#include <hip/hip_runtime.h>
#include <stdlib.h>

struct JobDescription{
  int JobID;
  int JobType;
  int numThreads;
  void *params;
};
typedef struct JobDescription *JobPointer; //needed to make these volatile

struct QueueRecord {
  JobPointer* Array; //Order matters here, we should improve this later
  int Capacity;
  int Rear;
  int Front;
  int ReadLock;
};

typedef struct QueueRecord *Queue;


////////////////////////////////////////////////////////////
// Locking Functions used to Sync warps access to Queues
////////////////////////////////////////////////////////////
__device__ void getLock(volatile Queue Q, volatile int *kill)
{
  while(atomicCAS(&(Q->ReadLock), 0, 1) != 0) if(*kill)return;
}

__device__ void releaseLock(volatile Queue Q)
{
  atomicExch(&(Q->ReadLock),0);
}

///////////////////////////////////////////////////////////
// Device Helper Functions
///////////////////////////////////////////////////////////

__device__ int d_IsEmpty(volatile Queue Q) {
  volatile int *rear = &(Q->Rear);
  return (*rear+1)%Q->Capacity == Q->Front;
}

__device__ int d_IsFull(volatile Queue Q) {
  volatile int *front = &(Q->Front);
  return (Q->Rear+2)%Q->Capacity == *front;
}


//////////////////////////////////////////////////////////
// Host Helper Functions
//////////////////////////////////////////////////////////
int h_IsEmpty(Queue Q) {
  return (Q->Rear+1)%Q->Capacity == Q->Front;
}

int h_IsFull(Queue Q) {
  return (Q->Rear+2)%Q->Capacity == Q->Front;
}

void *movePointer(void *p, int n){
   char * ret = (char *) p;
   return ((void *)(ret+n));
}

void printAnyErrors()
{
  hipError_t e = hipGetLastError();
  if(e!=hipSuccess){
    printf("CUDA Error: %s\n", hipGetErrorString( e ) );
  }
}



